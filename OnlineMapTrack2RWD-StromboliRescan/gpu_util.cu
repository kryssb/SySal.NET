#include "gpu_util.h"
#include "hip/hip_runtime.h"

namespace SySal {
namespace GPU {
int GetAvailableGPUs()
{
	hipError_t err;
	int count = 0;
	hipGetDeviceCount(&count);
	return count;
}
};
};